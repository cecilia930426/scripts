
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__device__ int d_value;

__global__ void test_Kernel()
{
	int threadID = threadIdx.x;

	d_value = 1;
	printf("gridDim %-3d, blockDim %-3d, blockIdx %-3d,threadID %-3d d_value%3d\n",gridDim.x,blockDim.x,blockIdx.x,threadID,d_value);
}
int main()
{
	int h_value = 0;

    dim3 blocks(8);         //gridDim = 8,blocksIdx,0-7
    dim3 threads(4);        //blockDim=4, threadIdx,0-3

    printf("Test blocks and threads ==============\n");

	test_Kernel<<<blocks,threads>>>();

    printf("use numbers ==============\n");

	//test_Kernel<<<8,4>>>();

	hipMemcpyFromSymbol(&h_value,HIP_SYMBOL(d_value),
		sizeof(int),0,hipMemcpyDeviceToHost);
	
	printf("Output from host: %d\n",h_value);
	return 0;
}

